#include <iostream>
#include <algorithm>

#include "common.h"

#define NUM_ELEMS 1024

struct CommData {
  hipIpcMemHandle_t memHandle;
  hipIpcEventHandle_t eventHandle;
};

int main() {
  int rank, world_size;
  // initializing MPI
  MPICHECK(MPI_Init(nullptr, nullptr));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &world_size));
  CUDACHECK(hipSetDevice(rank));

  float* host_buf = new float[NUM_ELEMS];
  float* device_buf, *peer_buf;
  CommData commData;
  hipEvent_t event;
  for (int i = 0; i <  1; i++) {
    if (rank == 0) {
      if (i == 0) {
        hipMalloc(&device_buf, NUM_ELEMS * sizeof(float));
      }
      hipMalloc(&device_buf, NUM_ELEMS * sizeof(float));

      std::fill(host_buf, host_buf + NUM_ELEMS, i + 1);
      CUDACHECK(hipMemcpy(device_buf, host_buf, NUM_ELEMS * sizeof(float),
                           hipMemcpyHostToDevice));
      hipDeviceSynchronize();
//      MPI_Barrier(MPI_COMM_WORLD);

      if (i == 0) {
        MPICHECK(MPI_Recv((void*)(&commData.memHandle),
                          sizeof(commData.memHandle), MPI_UNSIGNED_CHAR, 1, 0,
                          MPI_COMM_WORLD, MPI_STATUS_IGNORE));
        hipSetDevice(1);
        CUDACHECK(hipIpcOpenMemHandle((void**)&peer_buf, commData.memHandle,
                                       hipIpcMemLazyEnablePeerAccess));
        hipSetDevice(0);
        CUDACHECK(hipEventCreate(&event, hipEventDisableTiming |
                                          hipEventInterprocess));
        CUDACHECK(hipIpcGetEventHandle(
            (hipIpcEventHandle_t*)&commData.eventHandle, event));
        MPICHECK(MPI_Send((void*)(&commData.eventHandle),
                          sizeof(commData.eventHandle), MPI_UNSIGNED_CHAR, 1, 0,
                          MPI_COMM_WORLD));
      }

      printf("Pointer send to %p\n", peer_buf);
      CUDACHECK(hipMemcpy(peer_buf, device_buf, NUM_ELEMS * sizeof(float), hipMemcpyDefault));
//      CUDACHECK(hipMemcpyPeer(peer_buf, 1, device_buf, 0,
//                               NUM_ELEMS * sizeof(float)));
      hipEventRecord(event);
      hipEventSynchronize(event);
//      MPI_Barrier(MPI_COMM_WORLD);
      std::cout << rank << ". Source: " << host_buf[0] << " " << host_buf[NUM_ELEMS - 1] << std::endl;
    } else {
      if (i == 0) {
        hipMalloc(&peer_buf, NUM_ELEMS * sizeof(float));
      }
      std::fill(host_buf, host_buf + NUM_ELEMS, 0);
      CUDACHECK(hipMemcpy(peer_buf, host_buf, NUM_ELEMS * sizeof(float),
                           hipMemcpyHostToDevice));
      CUDACHECK(hipDeviceSynchronize());
//      MPI_Barrier(MPI_COMM_WORLD);

      if (i == 0) {
        CUDACHECK(hipIpcGetMemHandle(
            (&commData.memHandle), (void*)peer_buf));
        MPICHECK(MPI_Send((void*)(&commData.memHandle),
                          sizeof(commData.memHandle), MPI_UNSIGNED_CHAR, 0, 0,
                          MPI_COMM_WORLD));
        MPICHECK(MPI_Recv((void*)(&commData.eventHandle),
                          sizeof(commData.eventHandle), MPI_UNSIGNED_CHAR, 0, 0,
                          MPI_COMM_WORLD, MPI_STATUS_IGNORE));
        hipIpcOpenEventHandle(&event, commData.eventHandle);
      }

      hipEventSynchronize(event);
//      MPI_Barrier(MPI_COMM_WORLD);
      printf("Pointer recv %p\n", peer_buf);
      CUDACHECK(hipMemcpy(host_buf, peer_buf, NUM_ELEMS * sizeof(float),
                           hipMemcpyDeviceToHost));
      CUDACHECK(hipDeviceSynchronize());
      std::cout << rank << ". Result: " << host_buf[0] << " " << host_buf[NUM_ELEMS - 1] << std::endl;
    }
  }
  CUDACHECK(hipDeviceSynchronize());
//  if (rank == 0) {
//    hipSetDevice(1);
//    CUDACHECK(hipIpcCloseMemHandle(peer_buf));
//    hipSetDevice(0);
//    CUDACHECK(hipFree(device_buf));
//    CUDACHECK(hipEventDestroy(event));
//  } else {
//    CUDACHECK(hipFree(peer_buf));
//  }
  delete [] host_buf;
  MPICHECK(MPI_Finalize());
  return 0;
}